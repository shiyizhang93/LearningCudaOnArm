//
// Created by shiyi on 01/23/2022.
//


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

void __global__ add(const double *x, const double *y, double *z);
void check(const double *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(double) * N;
    // allocate host memory
    double *h_x = (double *) malloc(M);
    double *h_y = (double *) malloc(M);
    double *h_z = (double *) malloc(M);
    // initialize host x,y array
    for (int i = 0; i < N; ++i)
    {
        h_x[i] = a;
        h_y[i] = b;
    }

    // allocate device memory
    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);
    // copy data from host to device
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);
    // set grid size, block size and call add kernel function
    const int block_size = 128;
    const int grid_size = N / block_size;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z);
    // copy result back from device to host
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);
    // release host memory
    free(h_x);
    free(h_y);
    free(h_z);
    // release device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return 0;
}

void __global__ add(const double *x, const double *y, double *z)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    z[n] = x[n] + y[n];
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int i = 0; i < N; ++i)
    {
        if (fabs(z[i] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}
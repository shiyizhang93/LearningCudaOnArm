//! \Brief  Invoke device function by passing the value in reference method
//!
//! \Author Shiyi Zhang
//!
//! \Create 02/25/2022



#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

void __global__ add(const double *x, const double *y, double *z, int N);
void check(const double *z, const int N);

int main(void) {
    const int N = 100000001;
    const int M = sizeof(double) * N;
    // allocate host memory
    // initiate dynamic memory with new operator
    double *h_x = new double[N];
    double *h_y = new double[N];
    double *h_z = new double[N];

    // initialize host x,y array
    for (int i = 0; i < N; ++i) {
        h_x[i] = a;
        h_y[i] = b;
    }

    // allocate device memory
    double *d_x, *d_y, *d_z;
    hipMalloc((void **) &d_x, M);
    hipMalloc((void **) &d_y, M);
    hipMalloc((void **) &d_z, M);
    // copy data from host to device
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);
    // set grid size, block size and call add kernel function
    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size; // make grid_size round up to the closest int num
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    // copy result back from device to host
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);
    // release host memory
    // use delete operator to release
    delete[] h_x;
    delete[] h_y;
    delete[] h_z;
    // release device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return 0;
}

void __device__ add2_device(const double x, const double y, double &z)
{
    z = x + y;
}


void __global__ add(const double *x, const double *y, double *z, int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        add2_device(x[n], y[n], z[n]);
    }
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int i = 0; i < N; ++i)
    {
        if (fabs(z[i] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}



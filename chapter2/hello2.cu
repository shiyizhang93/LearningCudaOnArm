//
// Created by shiyi on 01/20/2022.
//


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGpu()
{
    printf("Hello World from the GPU!\n");
}

int main(void)
{
    helloFromGpu<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}

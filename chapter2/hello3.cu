//
// Created by shiyi on 01/21/2022.
//


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGpu()
{
    printf("Hello World from the GPU!\n");
}

int main(void)
{
    helloFromGpu<<<2, 4>>>(); // try with grid size 2 and block size 4
    hipDeviceSynchronize();

    return 0;
}

//
// Created by shiyi on 01/21/2022.
//


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGpu()
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("Hello World from block %d and thread %d!\n", bid, tid);
}

int main(void)
{
    helloFromGpu<<<2, 4>>>();
    hipDeviceSynchronize();

    return 0;
}
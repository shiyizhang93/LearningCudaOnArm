//
// Created by shiyi on 01/21/2022.
//


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGpu()
{
    const int bid = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    printf("Hello World from block-%d and thread-(%d, %d)!\n", bid, tx, ty);
}

int main(void)
{
    const dim3 block_size(2, 4);
    helloFromGpu<<<1, block_size>>>();
    hipDeviceSynchronize();

    return 0;
}